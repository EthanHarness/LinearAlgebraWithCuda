#include "hip/hip_runtime.h"
﻿#include "CMatrix.cuh"
#include <chrono>
#include <ctime>

#define ITERATIONS 1000

void CudaVNonCuda();
std::vector<std::pair<CMatrix, int>> readTestData();
std::vector<std::pair<CMatrix, int>> readTrainingData();

int main() {
    //CudaVNonCuda();
    std::vector<std::pair<CMatrix, int>> testData = readTestData();
    std::vector<std::pair<CMatrix, int>> trainData = readTrainingData();
    printCMatrix(testData[0].first);
    printCMatrix(trainData[0].first);
    return 0;
}

void CudaVNonCuda() {
    //Creates and sets a bunch of CMatrix's (Mainly for testing purposes)
    CMatrix CMatrixObj = createCMatrix(5, 5);
    CMatrix CMatrixObj1 = createCMatrix(5, 5);
    std::function<double(int, int)> foo; 
    std::function<double(int, int)> foo1;
    std::function<double(int, int)> foo2;

    foo = [](int x, int y) {
        return (double)((x * x) + y);
    };
    foo1 = [](int x, int y) {
        return (double)((y * y) + x);
    };
    foo2 = [](int x, int y) {
        return (double)(x + 1);
    };

    setCMatrix(foo1, CMatrixObj);
    setCMatrix(foo, CMatrixObj1);
    CMatrix CMatrixObj2 = CMatrixAdd(CMatrixObj, CMatrixObj1);

    //This does a bunch of Matrix multiplications.
    for(int i = 0; i < ITERATIONS; i+=10) {
        std::cout << "Iteration : " << i/10 << std::endl;
        std::cout << "Size of Matrix's are : " << i << "x" << i << std::endl;
        CMatrix m1 = createCMatrix(i, i);
        CMatrix m2 = createCMatrix(i, i);
        setCMatrix(foo, m1);
        setCMatrix(foo, m2);

        clock_t now = clock();
        CMatrix m3 = CMatrixMultiply(m1, m2);
        std::cout << "TIME: " << clock() - now << std::endl;
            
        now = clock();
        CMatrix m4 = multiply_cuda(m1, m2);
        std::cout << "TIME : " << clock() - now << std::endl << std::endl;
   }
}

std::vector<std::pair<CMatrix, int>> readTestData() {
    std::ifstream file("data/mnist_test.csv");

    if (!file.is_open()) {
        std::cerr << "Error: File could not be opened." << std::endl;
    }

    std::string line;
    std::vector<std::pair<CMatrix, int>> testData;
    
    //Need to consume the first line since its just header information
    std::getline(file, line);

    while (std::getline(file, line)) {
        std::stringstream ss(line);
        std::string value;

        std::vector<int> row;
        while (std::getline(ss, value, ',')) {
            int intValue = std::stoi(value);
            row.push_back(intValue);
        }

        int firstValue = row[0];
        row.erase(row.begin());

        CMatrix testingDataCMatrix = createCMatrix(1, 784);
        std::function<double(int, int)> foo;
        foo = [row](int x, int y) {
            return (double)(row[y]);
        };
        setCMatrix(foo, testingDataCMatrix);

        testData.push_back(std::make_pair(testingDataCMatrix, firstValue));
    }

    file.close();
    return testData;
}

std::vector<std::pair<CMatrix, int>> readTestData() {
    std::ifstream file("data/mnist_train.csv");

    if (!file.is_open()) {
        std::cerr << "Error: File could not be opened." << std::endl;
    }

    std::string line;
    std::vector<std::pair<CMatrix, int>> testData;
    
    //Need to consume the first line since its just header information
    std::getline(file, line);

    while (std::getline(file, line)) {
        std::stringstream ss(line);
        std::string value;

        std::vector<int> row;
        while (std::getline(ss, value, ',')) {
            int intValue = std::stoi(value);
            row.push_back(intValue);
        }

        int firstValue = row[0];
        row.erase(row.begin());

        CMatrix testingDataCMatrix = createCMatrix(1, 784);
        std::function<double(int, int)> foo;
        foo = [row](int x, int y) {
            return (double)(row[y]);
        };
        setCMatrix(foo, testingDataCMatrix);

        testData.push_back(std::make_pair(testingDataCMatrix, firstValue));
    }

    file.close();
    return testData;
}