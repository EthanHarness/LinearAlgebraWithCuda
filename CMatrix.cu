#include "hip/hip_runtime.h"
#include "CMatrix.cuh"

__global__ void multiplyWithCuda(CMatrix A, CMatrix B, CMatrix C) {

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	double sum = 0;

	for (int i = 0; i < A.width; ++i) {
		sum += A.elements[row * A.width + i] * B.elements[i * B.width + col];
	}
	C.elements[row * C.width + col] = sum;

};

__global__ void smultiplyWithCuda(CMatrix A, CMatrix B, double scalar) {

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	B.elements[row * B.width + col] = A.elements[row * A.width + col] * scalar;

};

__global__ void addWithCuda(CMatrix A, CMatrix B, CMatrix C) {

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	C.elements[row * C.width + col] = A.elements[row * A.width + col] + B.elements[row * B.width + col];

};

__global__ void sigmoidWithCuda(CMatrix A, CMatrix B) {

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	B.elements[row * B.width + col] = 1 / (1 + exp(A.elements[row * A.width + col]));

}

__global__ void tanhWithCuda(CMatrix A, CMatrix B) {

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	B.elements[row * B.width + col] = atanh(A.elements[row + A.width + col]);

}

__global__ void reluWithCuda(CMatrix A, CMatrix B) {

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	B.elements[row * B.width + col] = max(A.elements[row + A.width + col], 0.0);

}

__global__ void squareDiffWithCuda(CMatrix A, CMatrix B, CMatrix C) {

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	double diff = A.elements[row * A.width + col] - B.elements[row * B.width + col];
	C.elements[row * C.width + col] = pow(diff, 2);

}

void setCMatrix(std::function<double(int, int)> func, CMatrix res) {
	for (int i = 0; i < res.height; i++) {
		for (int j = 0; j < res.width; j++) {
			res.elements[i * res.width + j] = func(i, j);
		}
	}
}

void printCMatrix(CMatrix res) {

	for (int i = 0; i < res.height; i++) {
		for (int j = 0; j < res.width; j++) {
			std::cout << res.elements[i * res.width + j] << " ";
		}
		std::cout << "\n";
	}
}

CMatrix createCMatrix(int rows, int cols) {

	CMatrix res;
	res.height = rows;
	res.width = cols;
	res.elements = (double*)calloc(rows * cols, sizeof(double));
	return res;

}

CMatrix CMatrixAdd(CMatrix mat1, CMatrix mat2) {

	int cols = mat1.width;
	int rows = mat2.height;

	if (cols != mat2.width || rows != mat2.height)
		throw std::invalid_argument("Matricies are not the same size.");

	CMatrix result = createCMatrix(cols, rows);
	std::function<double(int, int)> add = [mat1, mat2, cols](int i, int j) {
		return mat1.elements[i * cols + j] + mat2.elements[i * cols + j];
	};

	setCMatrix(add, result);
	return result;

}

CMatrix CMatrixSMultiply(CMatrix mat, double scalar) {

	int cols = mat.width;
	int rows = mat.height;

	CMatrix res = createCMatrix(cols, rows);
	std::function<double(int, int)> smult = [mat, scalar](int i, int j) {
		return mat.elements[i * mat.width + j] * scalar;
	};
	setCMatrix(smult, res);
	return res;

}

CMatrix CMatrixMultiply(CMatrix mat1, CMatrix mat2) {

	int row1 = mat1.height;
	int row2 = mat2.height;
	int col1 = mat1.width;
	int col2 = mat2.width;

	if (col1 != row2)
		throw std::invalid_argument("Columns of matrix 1 do not equal the rows of matrix 2.");

	CMatrix res = createCMatrix(row1, col2);
	double* resHead = res.elements;
	double* mat1Head = mat1.elements;
	double* mat2Head = mat2.elements;

	for (int i = 0; i < row1; i++) {
		for (int j = 0; j < col2; j++) {
			for (int k = 0; k < row2; k++) {
				resHead[i * res.width + j] += mat1Head[i * col1 + k] * mat2Head[k * col2 + j];
			}
		}
	}

	return res;

}

CMatrix multiply_cuda(CMatrix mat1, CMatrix mat2) {

	int row1 = mat1.height;
	int row2 = mat2.height;
	int col1 = mat1.width;
	int col2 = mat2.width;

	if (col1 != row2)
		throw std::invalid_argument("Columns of matrix 1 do not equal the rows of matrix 2.");

	CMatrix res = createCMatrix(row1, col2);

	CMatrix device_matrix_A;
	device_matrix_A.width = mat1.width;
	device_matrix_A.height = mat1.height;
	size_t size = mat1.width * mat1.height * sizeof(double);
	hipMalloc(&device_matrix_A.elements, size);
	hipMemcpy(device_matrix_A.elements, mat1.elements, size, hipMemcpyHostToDevice);

	CMatrix device_matrix_B;
	device_matrix_B.width = mat2.width;
	device_matrix_B.height = mat2.height;
	size = mat2.width * mat2.height * sizeof(double);
	hipMalloc(&device_matrix_B.elements, size);
	hipMemcpy(device_matrix_B.elements, mat2.elements, size, hipMemcpyHostToDevice);

	CMatrix device_matrix_C;
	device_matrix_C.width = res.width;
	device_matrix_C.height = res.height;
	size = res.width * res.height * sizeof(double);
	hipMalloc(&device_matrix_C.elements, size);
	hipMemcpy(device_matrix_C.elements, res.elements, size, hipMemcpyHostToDevice);

	dim3 threadsPerBlock(row1, col2);
	dim3 numBlocks(1, 1);
	multiplyWithCuda << <numBlocks, threadsPerBlock >> > (device_matrix_A, device_matrix_B, device_matrix_C);
	hipDeviceSynchronize();

	hipMemcpy(res.elements, device_matrix_C.elements, size, hipMemcpyDeviceToHost);
	hipFree(device_matrix_A.elements);
	hipFree(device_matrix_B.elements);
	hipFree(device_matrix_C.elements);

	return res;

};

CMatrix smultiply_cuda(CMatrix mat, double scalar) {

	int rows = mat.height;
	int cols = mat.width;

	CMatrix res = createCMatrix(rows, cols);

	CMatrix device_matrix_A;
	device_matrix_A.width = mat.width;
	device_matrix_A.height = mat.height;
	size_t size = mat.width * mat.height * sizeof(double);
	hipMalloc(&device_matrix_A.elements, size);
	hipMemcpy(device_matrix_A.elements, mat.elements, size, hipMemcpyHostToDevice);

	CMatrix device_matrix_B;
	device_matrix_B.width = res.width;
	device_matrix_B.height = res.height;
	size = res.width * res.height * sizeof(double);
	hipMalloc(&device_matrix_B.elements, size);
	hipMemcpy(device_matrix_B.elements, res.elements, size, hipMemcpyHostToDevice);

	dim3 threadsPerBlock(rows, cols);
	dim3 numBlocks(1, 1);
	smultiplyWithCuda << <numBlocks, threadsPerBlock >> > (device_matrix_A, device_matrix_B, scalar);
	hipDeviceSynchronize();

	hipMemcpy(res.elements, device_matrix_B.elements, size, hipMemcpyDeviceToHost);
	hipFree(device_matrix_A.elements);
	hipFree(device_matrix_B.elements);

	return res;

}

CMatrix add_cuda(CMatrix mat1, CMatrix mat2) {

	int row1 = mat1.height;
	int row2 = mat2.height;
	int col1 = mat1.width;
	int col2 = mat2.width;

	if (col1 != col2 && row1 != row2)
		throw std::invalid_argument("Columns of matrix 1 do not equal the rows of matrix 2.");

	CMatrix res = createCMatrix(row1, col1);

	CMatrix device_matrix_A;
	device_matrix_A.width = mat1.width;
	device_matrix_A.height = mat1.height;
	size_t size = mat1.width * mat1.height * sizeof(double);
	hipMalloc(&device_matrix_A.elements, size);
	hipMemcpy(device_matrix_A.elements, mat1.elements, size, hipMemcpyHostToDevice);

	CMatrix device_matrix_B;
	device_matrix_B.width = mat2.width;
	device_matrix_B.height = mat2.height;
	size = mat2.width * mat2.height * sizeof(double);
	hipMalloc(&device_matrix_B.elements, size);
	hipMemcpy(device_matrix_B.elements, mat2.elements, size, hipMemcpyHostToDevice);

	CMatrix device_matrix_C;
	device_matrix_C.width = res.width;
	device_matrix_C.height = res.height;
	size = res.width * res.height * sizeof(double);
	hipMalloc(&device_matrix_C.elements, size);
	hipMemcpy(device_matrix_C.elements, res.elements, size, hipMemcpyHostToDevice);

	dim3 threadsPerBlock(row1, col1);
	dim3 numBlocks(1, 1);
	addWithCuda << <numBlocks, threadsPerBlock >> > (device_matrix_A, device_matrix_B, device_matrix_C);
	hipDeviceSynchronize();

	hipMemcpy(res.elements, device_matrix_B.elements, size, hipMemcpyDeviceToHost);
	hipFree(device_matrix_A.elements);
	hipFree(device_matrix_B.elements);
	hipFree(device_matrix_C.elements);

	return res;

}

CMatrix add_cuda(CMatrix mat1) {

	int row = mat1.height;
	int col = mat1.width;

	CMatrix res = createCMatrix(row, col);

	CMatrix device_matrix_A;
	device_matrix_A.width = mat1.width;
	device_matrix_A.height = mat1.height;
	size_t size = mat1.width * mat1.height * sizeof(double);
	hipMalloc(&device_matrix_A.elements, size);
	hipMemcpy(device_matrix_A.elements, mat1.elements, size, hipMemcpyHostToDevice);

	CMatrix device_matrix_B;
	device_matrix_B.width = res.width;
	device_matrix_B.height = res.height;
	size = res.width * res.height * sizeof(double);
	hipMalloc(&device_matrix_B.elements, size);
	hipMemcpy(device_matrix_B.elements, res.elements, size, hipMemcpyHostToDevice);

	dim3 threadsPerBlock(row, col);
	dim3 numBlocks(1, 1);
	sigmoidWithCuda << <numBlocks, threadsPerBlock >> > (device_matrix_A, device_matrix_B, device_matrix_C);
	hipDeviceSynchronize();

	hipMemcpy(res.elements, device_matrix_B.elements, size, hipMemcpyDeviceToHost);
	hipFree(device_matrix_A.elements);
	hipFree(device_matrix_B.elements);

	return res;

}

CMatrix relu_cuda(CMatrix mat1) {

	int row = mat1.height;
	int col = mat1.width;

	CMatrix res = createCMatrix(row, col);

	CMatrix device_matrix_A;
	device_matrix_A.width = mat1.width;
	device_matrix_A.height = mat1.height;
	size_t size = mat1.width * mat1.height * sizeof(double);
	hipMalloc(&device_matrix_A.elements, size);
	hipMemcpy(device_matrix_A.elements, mat1.elements, size, hipMemcpyHostToDevice);

	CMatrix device_matrix_B;
	device_matrix_B.width = res.width;
	device_matrix_B.height = res.height;
	size = res.width * res.height * sizeof(double);
	hipMalloc(&device_matrix_B.elements, size);
	hipMemcpy(device_matrix_B.elements, res.elements, size, hipMemcpyHostToDevice);

	dim3 threadsPerBlock(row, col);
	dim3 numBlocks(1, 1);
	reluWithCuda << <numBlocks, threadsPerBlock >> > (device_matrix_A, device_matrix_B, device_matrix_C);
	hipDeviceSynchronize();

	hipMemcpy(res.elements, device_matrix_B.elements, size, hipMemcpyDeviceToHost);
	hipFree(device_matrix_A.elements);
	hipFree(device_matrix_B.elements);

	return res;

}

CMatrix tanh_cuda(CMatrix mat1) {

	int row = mat1.height;
	int col = mat1.width;

	CMatrix res = createCMatrix(row, col);

	CMatrix device_matrix_A;
	device_matrix_A.width = mat1.width;
	device_matrix_A.height = mat1.height;
	size_t size = mat1.width * mat1.height * sizeof(double);
	hipMalloc(&device_matrix_A.elements, size);
	hipMemcpy(device_matrix_A.elements, mat1.elements, size, hipMemcpyHostToDevice);

	CMatrix device_matrix_B;
	device_matrix_B.width = res.width;
	device_matrix_B.height = res.height;
	size = res.width * res.height * sizeof(double);
	hipMalloc(&device_matrix_B.elements, size);
	hipMemcpy(device_matrix_B.elements, res.elements, size, hipMemcpyHostToDevice);

	dim3 threadsPerBlock(row, col);
	dim3 numBlocks(1, 1);
	tanhWithCuda << <numBlocks, threadsPerBlock >> > (device_matrix_A, device_matrix_B, device_matrix_C);
	hipDeviceSynchronize();

	hipMemcpy(res.elements, device_matrix_B.elements, size, hipMemcpyDeviceToHost);
	hipFree(device_matrix_A.elements);
	hipFree(device_matrix_B.elements);

	return res;

}

CMatrix computeLossMatrix_cuda(CMatrix computedMatrix, CMatrix expectedMatrix) {

	int row = computedMatrix.height;
	int col = computedMatrix.width;

	if (expectedMatrix.height != row && expectedMatrix.width != col)
		throw std::invalid_argument("Matricies are not of equal size");

	CMatrix res = createCMatrix(row, col);

	CMatrix device_matrix_A;
	device_matrix_A.width = computedMatrix.width;
	device_matrix_A.height = computedMatrix.height;
	size_t size = computedMatrix.width * computedMatrix.height * sizeof(double);
	hipMalloc(&device_matrix_A.elements, size);
	hipMemcpy(device_matrix_A.elements, computedMatrix.elements, size, hipMemcpyHostToDevice);

	CMatrix device_matrix_B;
	device_matrix_B.width = expectedMatrix.width;
	device_matrix_B.height = expectedMatrix.height;
	size = expectedMatrix.width * expectedMatrix.height * sizeof(double);
	hipMalloc(&device_matrix_B.elements, size);
	hipMemcpy(device_matrix_B.elements, expectedMatrix.elements, size, hipMemcpyHostToDevice);

	CMatrix device_matrix_C;
	device_matrix_C.width = res.width;
	device_matrix_C.height = res.height;
	size = res.width * res.height * sizeof(double);
	hipMalloc(&device_matrix_C.elements, size);
	hipMemcpy(device_matrix_C.elements, res.elements, size, hipMemcpyHostToDevice);

	dim3 threadsPerBlock(row, col);
	dim3 numBlocks(1, 1);
	squareDiffWithCuda << <numBlocks, threadsPerBlock >> > (device_matrix_A, device_matrix_B, device_matrix_C);
	hipDeviceSynchronize();

	hipMemcpy(res.elements, device_matrix_C.elements, size, hipMemcpyDeviceToHost);
	hipFree(device_matrix_A.elements);
	hipFree(device_matrix_B.elements);
	hipFree(device_matrix_C.elements);

	return res;

}